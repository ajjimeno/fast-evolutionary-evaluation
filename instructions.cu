#ifndef PROGRAM_INSTRUCTIONS_C
#define PROGRAM_INSTRUCTIONS_C

#include "types.cuh"

__device__ int get_0(Run *run, Program **)
{
    return 0;
}
__device__ int get_1(Run *run, Program **)
{
    return 1;
}
__device__ int get_2(Run *run, Program **)
{
    return 2;
}
__device__ int get_3(Run *run, Program **)
{
    return 3;
}
__device__ int get_4(Run *run, Program **)
{
    return 4;
}
__device__ int get_5(Run *run, Program **)
{
    return 5;
}
__device__ int get_6(Run *run, Program **)
{
    return 6;
}
__device__ int get_7(Run *run, Program **)
{
    return 7;
}
__device__ int get_8(Run *run, Program **)
{
    return 8;
}
__device__ int get_9(Run *run, Program **)
{
    return 9;
}

__device__ int testing_input_beginning(Run *run, Program **)
{
    return run->input_x == 0;
}

__device__ int testing_input_read(Run *run, Program **)
{
    Problem p = run->problem;
    int **i = p.input;
    return i[run->input_x][run->input_y];
}

__device__ int testing_output_write(Run *run, Program **p)
{
    // int value = (p[0]->pointer)(run, p[0]->args);
    run->output[run->output_y][run->output_x] = 0; // value;

    return 0;
}

typedef int (*pfunc)(Run *run, Program **p);

__device__ pfunc dev_func_ptr = get_0;

__device__ pfunc dev_func_ptr_write = testing_output_write;


Program *copy_to_device(Program *p)
{
    Program *h_p = (Program *)malloc(sizeof(Program));

    h_p->n_args = p->n_args;

    if (p->n_args>0)
    {
        Program ** args = (Program **)malloc(p->n_args*sizeof(Program *));
        // Save the subprograms in device memory
        for (int i = 0; i < p->n_args; i++)
        {
            args[i] = copy_to_device(p->args[i]);
        }

        hipMalloc(&h_p->args, p->n_args*sizeof(Program *));
        hipMemcpy(h_p->args, args, p->n_args*sizeof(Program *), hipMemcpyHostToDevice);

        free(args);
    }

    Program * p_p;

    h_p->pointer = p->pointer;

    hipMalloc(&p_p, sizeof(Program));
    hipMemcpy(p_p, h_p, sizeof(Program), hipMemcpyHostToDevice);

    return p_p;
}

Program *copy_programs_to_gpu(int n_programs)
{

    // Example program creation
    Program program_get = {
        NULL, // progran
        0,            // arguments
        NULL          // pointer to args
    };

    hipMemcpyFromSymbol(&program_get.pointer, HIP_SYMBOL(dev_func_ptr), sizeof(pfunc));

    Program **subprogram = new struct Program *[1];
    subprogram[0] = &program_get;

    // Example program creation
    Program program = {
        NULL, // progran
        1,                  // arguments
        subprogram          // pointer to args
    };

    hipMemcpyFromSymbol(&program.pointer, HIP_SYMBOL(dev_func_ptr_write), sizeof(pfunc));

    // Copy programs to device memory
    Program * d_p = copy_to_device(&program);

    // Create array of programs in host memory
    Program *programs = (Program *)malloc(n_programs * sizeof(struct Problem));

    for (int i = 0; i < n_programs; i++)
    {
        hipMemcpy(&programs[i], d_p, sizeof(Program), hipMemcpyDeviceToHost);
    }

    // Copy array of programs from host memory to device memory
    Program *d_programs;

    hipMalloc(&d_programs, n_programs * sizeof(struct Problem));

    hipMemcpy(d_programs, programs, n_programs * sizeof(struct Problem), hipMemcpyHostToDevice);

    free(programs);
    
    return d_programs;
}

#endif