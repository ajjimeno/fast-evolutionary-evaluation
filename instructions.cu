#include "hip/hip_runtime.h"
#ifndef PROGRAM_INSTRUCTIONS_C
#define PROGRAM_INSTRUCTIONS_C

#include "types.cuh"
#include <thread>
#include <unordered_map>
#include <vector>

__device__ int get0(Run *run, int *)
{
    return 0;
}
__device__ int get1(Run *run, int *)
{
    return 1;
}
__device__ int get2(Run *run, int *)
{
    return 2;
}
__device__ int get3(Run *run, int *)
{
    return 3;
}
__device__ int get4(Run *run, int *)
{
    return 4;
}
__device__ int get5(Run *run, int *)
{
    return 5;
}
__device__ int get6(Run *run, int *)
{
    return 6;
}
__device__ int get7(Run *run, int *)
{
    return 7;
}
__device__ int get8(Run *run, int *)
{
    return 8;
}
__device__ int get9(Run *run, int *)
{
    return 9;
}

__device__ int get_testing_length_input_x(Run *run, int *)
{
    return run->problem.input_x;
}

__device__ int get_testing_length_input_y(Run *run, int *)
{
    return run->problem.input_y;
}

__device__ int get_testing_length_output_x(Run *run, int *)
{
    return run->problem.output_x;
}

__device__ int get_testing_length_output_y(Run *run, int *)
{
    return run->problem.output_y;
}

__device__ int get_testing_input_position_y(Run *run, int *)
{
    return run->input_y;
}

__device__ int get_testing_input_position_x(Run *run, int *)
{
    return run->input_x;
}

__device__ int get_testing_output_position_y(Run *run, int *)
{
    return run->output_y;
}

__device__ int get_testing_output_position_x(Run *run, int *)
{
    return run->output_x;
}

__device__ int testing_input_max(Run *run, int *)
{
    int *arr = run->problem.input[run->input_y];
    int max = arr[0];
    for (int i = 1; i < run->problem.output_x; i++)
    {
        if (arr[i] > max)
        {
            max = arr[i];
        }
    }
    return max;
}

__device__ int testing_input_min(Run *run, int *)
{
    int *arr = run->problem.input[run->input_y];
    int min = arr[0];
    for (int i = 1; i < run->problem.output_x; i++)
    {
        if (arr[i] < min)
        {
            min = arr[i];
        }
    }
    return min;
}

__device__ int testing_input_read(Run *run, int *)
{
    return run->problem.input[run->input_y][run->input_x];
}

__device__ int testing_output_read_previous(Run *run, int *)
{
    if (run->output_x > 0)
        return run->output[run->output_y][run->output_x - 1];

    return -1;
}

__device__ int testing_output_read(Run *run, int *)
{
    return run->output[run->output_y][run->output_x];
}

__device__ int testing_reset_input_position(Run *run, int *)
{
    run->input_x = 0;
    return 0;
}

__device__ int testing_reset_input_down_position(Run *run, int *)
{
    run->input_y = 0;
    return 0;
}

__device__ int testing_output_write_previous(Run *run, int *p)
{

    if (run->output_x > 0)
    {
        int node_id = run->program_offset + p[0];
        Node node = run->programs->nodes[node_id];

        int value = run->pfuncs[(node.pointer)](run, node.args);
        run->output[run->output_y][run->output_x - 1] = value;
    }

    return 0;
}

__device__ int testing_output_write(Run *run, int *p)
{
    int node_id = run->program_offset + p[0];
    Node node = run->programs->nodes[node_id];
    int value = run->pfuncs[(node.pointer)](run, node.args);
    run->output[run->output_y][run->output_x] = value;

    return 0;
}

__device__ int testing_reset_output_position(Run *run, int *)
{
    run->output_x = 0;
    return 0;
}

__device__ int testing_reset_output_down_position(Run *run, int *)
{
    run->output_y = 0;
    return 0;
}

__device__ int testing_output_move_left(Run *run, int *)
{
    if (run->output_x > 0)
        run->output_x--;

    return 0;
}

__device__ int testing_output_move_right(Run *run, int *)
{
    if (run->output_x < (run->problem.output_x - 1))
        run->output_x++;

    return 0;
}

__device__ int testing_output_move_down(Run *run, int *)
{
    if (run->output_y < (run->problem.output_y - 1))
        run->output_y++;
    // else
    //     status = -1;
    return 0;
}

__device__ int testing_output_move_up(Run *run, int *)
{
    if (run->output_y > 0)
        run->output_y--;

    return 0;
}

__device__ int testing_is_output_end(Run *run, int *)
{
    return run->output_x == (run->problem.output_x - 1);
}

__device__ int testing_is_output_down(Run *run, int *)
{
    return run->output_y == (run->problem.output_y - 1);
}

__device__ int testing_input_move_left(Run *run, int *)
{
    if (run->input_x > 0)
        run->input_x--;

    return 0;
}

__device__ int testing_input_move_right(Run *run, int *)
{
    if (run->input_x < (run->problem.input_x - 1))
        run->input_x++;

    return 0;
}

__device__ int testing_input_move_down(Run *run, int *)
{
    if (run->input_y < (run->problem.input_y - 1))
        run->input_y++;

    return 0;
}

__device__ int testing_input_move_up(Run *run, int *)
{
    if (run->input_y > 0)
        run->input_y--;

    return 0;
}

__device__ int comparison(Run *run, int *p)
{
    int node_0_id = run->program_offset + p[0];
    Node node_0 = run->programs->nodes[node_0_id];

    if (run->pfuncs[(node_0.pointer)](run, node_0.args) == 1)
    {
        int node_1_id = run->program_offset + p[1];
        Node node_1 = run->programs->nodes[node_1_id];
        run->pfuncs[(node_1.pointer)](run, node_1.args);
    }
    else
    {
        int node_2_id = run->program_offset + p[2];
        Node node_2 = run->programs->nodes[node_2_id];
        run->pfuncs[(node_2.pointer)](run, node_2.args);
    }

    return 0;
}

__device__ int bigger_than_output_next(Run *run, int *p)
{
    /*
    if (output_position_x < (get_length_output_x(NULL) - 1))
    {
        return data->training.at(input_id).at(1).array[output_position_y][output_position_x] >
               data->training.at(input_id).at(1).array[output_position_y][output_position_x + 1];
    }*/

    return 0;
}

__device__ int bigger_than_testing_output_next(Run *run, int *)
{
    if (run->output_x < (run->problem.output_x - 1))
    {
        return run->output[run->output_y][run->output_x] >
               run->output[run->output_y][run->output_x + 1];
    }

    return 0;
}

__device__ int swap_testing_output_next(Run *run, int *)
{
    if (!testing_is_output_end(run, NULL))
    {
        int tmp = run->output[run->output_y][run->output_x];
        run->output[run->output_y][run->output_x] = run->output[run->output_y][run->output_x + 1];
        run->output[run->output_y][run->output_x + 1] = tmp;
    }

    return 0;
}

__device__ int bigger_than(Run *run, int *p)
{
    int node_0_id = run->program_offset + p[0];
    Node node_0 = run->programs->nodes[node_0_id];
    int output1 = run->pfuncs[(node_0.pointer)](run, node_0.args);

    int node_1_id = run->program_offset + p[1];
    Node node_1 = run->programs->nodes[node_1_id];
    int output2 = run->pfuncs[(node_1.pointer)](run, node_1.args);
    return output1 > output2;
}

__device__ int equal(Run *run, int *p)
{
    int node_0_id = run->program_offset + p[0];
    Node node_0 = run->programs->nodes[node_0_id];
    int output1 = run->pfuncs[(node_0.pointer)](run, node_0.args);

    int node_1_id = run->program_offset + p[1];
    Node node_1 = run->programs->nodes[node_1_id];
    int output2 = run->pfuncs[(node_1.pointer)](run, node_1.args);
    return output1 == output2;
}

__device__ int no(Run *run, int *p)
{
    int node_0_id = run->program_offset + p[0];
    Node node_0 = run->programs->nodes[node_0_id];
    int output1 = run->pfuncs[(node_0.pointer)](run, node_0.args);
    return !output1;
}

__device__ int prog2(Run *run, int *p)
{
    int node_0_id = run->program_offset + p[0];
    Node node_0 = run->programs->nodes[node_0_id];
    run->pfuncs[(node_0.pointer)](run, node_0.args);

    int node_1_id = run->program_offset + p[1];
    Node node_1 = run->programs->nodes[node_1_id];
    run->pfuncs[(node_1.pointer)](run, node_1.args);
    return 0;
}

__device__ int prog3(Run *run, int *p)
{

    int node_0_id = run->program_offset + p[0];
    Node node_0 = run->programs->nodes[node_0_id];
    run->pfuncs[(node_0.pointer)](run, node_0.args);

    int node_1_id = run->program_offset + p[1];
    Node node_1 = run->programs->nodes[node_1_id];
    run->pfuncs[(node_1.pointer)](run, node_1.args);

    int node_2_id = run->program_offset + p[2];
    Node node_2 = run->programs->nodes[node_2_id];
    run->pfuncs[(node_2.pointer)](run, node_2.args);

    return 0;
}

int loop(Run *run, int *p)
{
    int node_0_id = run->program_offset + p[0];
    Node node_0 = run->programs->nodes[node_0_id];
    int v = run->pfuncs[(node_0.pointer)](run, node_0.args);

    run->inner_loop++;

    if (run->inner_loop < 5 && v > 0 && v <= 30)
    {
        for (int i = 0; i < v; i++)
        {
            int node_1_id = run->program_offset + p[1];
            Node node_1 = run->programs->nodes[node_1_id];
            run->pfuncs[(node_1.pointer)](run, node_1.args);
        }
    }
    else
    {
        run->status = -2;
    }

    run->inner_loop--;

    return 0;
}

__device__ int dowhile(Run *run, int *p)
{
    int c = 0;

    run->inner_loop++;

    if (run->inner_loop < 3)
    {
        int node_0_id = run->program_offset + p[0];
        Node node_0 = run->programs->nodes[node_0_id];
        while (!run->pfuncs[(node_0.pointer)](run, node_0.args) && c < 10)
        {
            c++;
            int node_1_id = run->program_offset + p[1];
            Node node_1 = run->programs->nodes[node_1_id];
            run->pfuncs[(node_1.pointer)](run, node_1.args);
        }
    }
    else
    {
        run->status = -2;
    }

    run->inner_loop--;

    return 0;
}

__device__ int read_memory(Run *run, int *)
{
    return run->memory;
}

__device__ int write_memory(Run *run, int *p)
{
    int node_0_id = run->program_offset + p[0];
    Node node_0 = run->programs->nodes[node_0_id];
    int value = run->pfuncs[(node_0.pointer)](run, node_0.args);

    run->memory = value;

    return 0;
}

#define MAP_INSTRUCTIONS std::unordered_map<std::string, int>

__global__ void fill_function_pointers(pfunc *pfuncs)
{
    pfuncs[0] = get0;
    pfuncs[1] = get1;
    pfuncs[2] = get2;
    pfuncs[3] = get3;
    pfuncs[4] = get4;
    pfuncs[5] = get5;
    pfuncs[6] = get6;
    pfuncs[7] = get7;
    pfuncs[8] = get8;
    pfuncs[9] = get9;
    pfuncs[10] = get_testing_length_input_x;
    pfuncs[11] = get_testing_length_input_y;
    pfuncs[12] = get_testing_length_output_x;
    pfuncs[13] = get_testing_length_output_y;
    pfuncs[14] = get_testing_input_position_y;
    pfuncs[15] = get_testing_input_position_x;
    pfuncs[16] = get_testing_output_position_y;
    pfuncs[17] = get_testing_output_position_x;
    pfuncs[18] = testing_input_max;
    pfuncs[19] = testing_input_min;
    pfuncs[20] = testing_input_read;
    pfuncs[21] = testing_output_read_previous;
    pfuncs[22] = testing_output_read;
    pfuncs[23] = testing_reset_input_position;
    pfuncs[24] = testing_reset_input_down_position;
    pfuncs[25] = testing_output_write_previous;
    pfuncs[26] = testing_output_write;
    pfuncs[27] = testing_reset_output_position;
    pfuncs[28] = testing_reset_output_down_position;
    pfuncs[29] = testing_output_move_left;
    pfuncs[30] = testing_output_move_right;
    pfuncs[31] = testing_output_move_down;
    pfuncs[32] = testing_output_move_up;
    pfuncs[33] = testing_is_output_end;
    pfuncs[34] = testing_is_output_down;
    pfuncs[35] = testing_input_move_left;
    pfuncs[36] = testing_input_move_right;
    pfuncs[37] = testing_input_move_down;
    pfuncs[38] = testing_input_move_up;
    pfuncs[39] = comparison;
    pfuncs[40] = bigger_than_output_next;
    pfuncs[41] = bigger_than_testing_output_next;
    pfuncs[42] = swap_testing_output_next;
    pfuncs[43] = bigger_than;
    pfuncs[44] = equal;
    pfuncs[45] = no;
    pfuncs[46] = prog2;
    pfuncs[47] = prog3;
    pfuncs[48] = dowhile;
    pfuncs[49] = read_memory;
    pfuncs[50] = write_memory;
}

MAP_INSTRUCTIONS get_map()
{

    MAP_INSTRUCTIONS map;

    map["get0"] = 0;
    map["get1"] = 1;
    map["get2"] = 2;
    map["get3"] = 3;
    map["get4"] = 4;
    map["get5"] = 5;
    map["get6"] = 6;
    map["get7"] = 7;
    map["get8"] = 8;
    map["get9"] = 9;
    map["get_testing_length_input_x"] = 10;
    map["get_testing_length_input_y"] = 11;
    map["get_testing_length_output_x"] = 12;
    map["get_testing_length_output_y"] = 13;
    map["get_testing_input_position_y"] = 14;
    map["get_testing_input_position_x"] = 15;
    map["get_testing_output_position_y"] = 16;
    map["get_testing_output_position_x"] = 17;
    map["testing_input_max"] = 18;
    map["testing_input_min"] = 19;
    map["testing_input_read"] = 20;
    map["testing_output_read_previous"] = 21;
    map["testing_output_read"] = 22;
    map["testing_reset_input_position"] = 23;
    map["testing_reset_input_down_position"] = 24;
    map["testing_output_write_previous"] = 25;
    map["testing_output_write"] = 26;
    map["testing_reset_output_position"] = 27;
    map["testing_reset_output_down_position"] = 28;
    map["testing_output_move_left"] = 29;
    map["testing_output_move_right"] = 30;
    map["testing_output_move_down"] = 31;
    map["testing_output_move_up"] = 32;
    map["testing_is_output_end"] = 33;
    map["testing_is_output_down"] = 34;
    map["testing_input_move_left"] = 35;
    map["testing_input_move_right"] = 36;
    map["testing_input_move_down"] = 37;
    map["testing_input_move_up"] = 38;
    map["comparison"] = 39;
    map["bigger_than_output_next"] = 40;
    map["bigger_than_testing_output_next"] = 41;
    map["swap_testing_output_next"] = 42;
    map["bigger_than"] = 43;
    map["equal"] = 44;
    map["no"] = 45;
    map["prog2"] = 46;
    map["prog3"] = 47;
    map["dowhile"] = 48;
    map["read_memory"] = 49;
    map["write_memory"] = 50;

    return map;
}

int getProgram(std::string string, MAP_INSTRUCTIONS map, std::vector<Node> *nodes, int &position)
{
    int program = -1;

    std::vector<int> subprograms;

    int initial_position = position;

    while (initial_position < string.length() && string[initial_position] == ' ')
        initial_position++;

    // Traverse string into tree
    while (position < string.length())
    {
        if (string[position] == '(')
        {
            // Create new program entry
            program = nodes->size();
            int pointer = map[string.substr(initial_position, position - initial_position)];

            nodes->push_back({pointer, 0, {0, 0, 0}});

            position++;

            if (string[position] == ')')
            {
                position++;
                return program;
            }
            else
            {
                int sub = getProgram(string, map, nodes, position);
                subprograms.push_back(sub);
            }
        }
        else if (string[position] == ')')
        {
            // Close and return
            position++;
            nodes->at(program).n_args = subprograms.size();

            for (int i = 0; i < subprograms.size(); i++)
            {
                nodes->at(program).args[i] = subprograms[i];
            }

            return program;
        }
        else if (string[position] == ',')
        {
            // After this, there is a new program
            position++;
            subprograms.push_back(getProgram(string, map, nodes, position));
        }
        else
        {
            position++;
        }
    }

    return program;
}

void getProgram(std::string string, MAP_INSTRUCTIONS map, std::vector<Node> *nodes)
{
    int position = 0;
    getProgram(string, map, nodes, position);
}

void copy_program(int start_index, int end_index, std::vector<int> *programs, std::vector<Node> *nodes, std::string *code, MAP_INSTRUCTIONS map)
{
    for (int i = start_index; i < end_index; ++i)
    {
        std::vector<Node> subnodes;
        getProgram(code[i], map, &subnodes);

        programs->push_back(nodes->size());

        for (int i = 0; i < subnodes.size(); i++)
        {
            nodes->push_back({subnodes[i].pointer, subnodes[i].n_args, {subnodes[i].args[0], subnodes[i].args[1], subnodes[i].args[2]}});
        }
    }
}

Programs *copy_programs_to_gpu(int n_programs, std::string *code)
{
    MAP_INSTRUCTIONS map = get_map();

    // Create array of programs in host memory
    Programs *d_sprograms;
    hipMallocManaged((void **)&d_sprograms, sizeof(struct Problems));

    int n_threads = std::min(n_programs, 1);
    int chunk_size = n_programs / n_threads;

    std::vector<std::thread> threads;

    std::vector<int> programs;
    std::vector<Node> nodes;
    for (int i = 0; i < n_threads; ++i)
    {
        int start_index = i * chunk_size;
        int end_index = (i == n_threads - 1) ? n_programs : (i + 1) * chunk_size;

        threads.emplace_back(copy_program, start_index, end_index, &programs, &nodes, code, map);
    }

    for (auto &t : threads)
    {
        t.join();
    }

    d_sprograms->n_nodes = nodes.size();
    d_sprograms->n_programs = programs.size();

    hipMalloc(&d_sprograms->nodes, nodes.size() * sizeof(Node));
    hipMemcpy(d_sprograms->nodes, nodes.data(), nodes.size() * sizeof(Node), hipMemcpyHostToDevice);

    hipMalloc(&d_sprograms->programs, programs.size() * sizeof(int));
    hipMemcpy(d_sprograms->programs, programs.data(), programs.size() * sizeof(int), hipMemcpyHostToDevice);

    return d_sprograms;
}

int free_programs_from_gpu(Programs *programs)
{
    hipFree(programs->nodes);
    hipFree(programs->programs);
    hipFree(programs);
    return 0;
}

#endif