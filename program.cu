#include "hip/hip_runtime.h"
#include <iostream>
#include "instructions.cu"
#include "types.cuh"

#ifndef PROGRAM_RUNNING
#define PROGRAM_RUNNING

#define N_BLOCKS 500
#define N_THREADS 1024



// Programs, Problems, split programs
__global__ void create_and_run(Programs *programs, int n_programs, Instances *problems, float *accuracy, int blocks, int threads)
{
	int programs_per_block = (n_programs / (blocks * threads)) + 1;

	int start = (blockIdx.x * blockDim.x + threadIdx.x) * programs_per_block;
	int end = start + programs_per_block;

	for (int i = start; i < end && i < n_programs; i++)
	{
		accuracy[i] = run(programs, i, problems);
	}
}

int execute_and_evaluate(int n_programs, STRING **programs, float *accuracy, Instances *problems)
{
	hipError_t err;
	float *d_accuracy;

	hipMallocManaged(&d_accuracy, n_programs * sizeof(float));

	Programs *d_programs = copy_programs_to_gpu(n_programs, programs);

	err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("Error creating programs: %s\n", hipGetErrorString(err));
		return 1;
	}

	std::cout << "Starting kernel " << n_programs << std::endl;

	int threads = std::min(N_THREADS, n_programs);

	int blocks = std::min((int)(n_programs / threads), N_BLOCKS);

	create_and_run<<<blocks, threads>>>(d_programs, n_programs, problems, d_accuracy, blocks, threads);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("Error launching kernel: %s\n", hipGetErrorString(err));
		// Handle the error (e.g., exit the program)
		return 1;
	}
	std::cout << "Kernel finished" << std::endl;

	hipMemcpy(accuracy, d_accuracy, n_programs * sizeof(float), hipMemcpyDeviceToHost);

	err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("Error launching kernel: %s\n", hipGetErrorString(err));
		// Handle the error (e.g., exit the program)
		return 1;
	}

	float total = 0.0;

	for (int i = 0; i < n_programs; i++)
	{
		total += accuracy[i];
	}

	std::cout << "Total: " << total << " " << n_programs << std::endl;

	hipFree(d_accuracy);

	free_programs_from_gpu(d_programs);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("Error freeing programs: %s\n", hipGetErrorString(err));
		return 1;
	}

	return 0;
}

/*
int main()
{
	int n_programs = 30000;
	std::vector<std::string> programs;

	for (int i = 0; i < n_programs; i++)
	{
		programs.push_back("prog2(testing_output_write(get0()),testing_output_move_right())");
		// programs.push_back("get0()");
	}

	float *accuracy;
	accuracy = (float *)malloc(n_programs * sizeof(float));

	execute_and_evaluate(n_programs, programs.data(), accuracy);
}*/

#endif