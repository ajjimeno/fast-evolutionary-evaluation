#include "hip/hip_runtime.h"
#include <iostream>
#include "instructions.cu"
#include "problems.cu"
#include "types.cuh"

#define N_BLOCKS 50
#define N_THREADS 100

__device__ float run(Programs *programs, int program_id, Instances *problems, pfunc *pfuncs)
{
	float total_accuracy = 0.0;

	for (int p = 0; p < problems->n_instances; p++)
	{
		int **output = (int **)malloc(problems->instances[p].output.y * sizeof(int *));

		for (int i = 0; i < problems->instances[p].output.y; i++)
		{
			output[i] = (int *)malloc(problems->instances[p].output.x * sizeof(int));
			for (int j = 0; j < problems->instances[p].output.x; j++)
			{
				output[i][j] = problems->instances[p].output.array[i][j];
			}
		}

		Run *r = (Run *)malloc(sizeof(Run));

		r[0] = {
			0,								// input_x
			0,								// input_y
			0,								// output_x
			0,								// output_y
			pfuncs,							// funcs
			problems->instances[p],			// problem
			output,							// output
			0,								// inner_loop
			0,								// status
			0,								// memory
			programs,						// programs
			programs->programs[program_id], // program_offset
			0,								// training_id
			0,								// training_input_x
			0,								// training_input_y
			0,								// training_output_x
			0								// training_output_y
		};

		for (int i = 0; i < 100; i++)
		{
			Node node = programs->nodes[programs->programs[program_id]];
			pfuncs[node.pointer](r, node.args);
		}

		total_accuracy += accuracy_calculation(problems->instances[p], output);

		free(r);

		for (int i = 0; i < problems->instances[p].output.y; i++)
		{
			free(output[i]);
		}
		free(output);
	}
	return total_accuracy / (float)problems->n_instances;
}

// Programs, Problems, split programs
__global__ void create_and_run(Programs *programs, int n_programs, Instances *problems, pfunc *pfuncs, float *accuracy, int blocks, int threads)
{
	int programs_per_block = n_programs / (blocks * threads);

	int start = (blockIdx.x * blockDim.x + threadIdx.x) * programs_per_block;
	int end = start + programs_per_block;

	for (int i = start; i < end; i++)
	{
		if (i < n_programs)
			accuracy[i] = run(programs, i, problems, pfuncs);
	}
}

int execute_and_evaluate(int n_programs, std::string *programs, float *accuracy, Instances *problems)
{
	int device_count = 0;
	// Get the number of CUDA-capable devices
	hipError_t err = hipGetDeviceCount(&device_count);

	if (err != hipSuccess)
	{
		std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
		return 1;
	}
	else
	{
		std::cout << "Device count: " << device_count << std::endl;
	}

	float *d_accuracy;

	hipMallocManaged(&d_accuracy, n_programs * sizeof(float));

	pfunc *d_pfuncs;
	hipMallocManaged(&d_pfuncs, 2 * sizeof(pfunc));

	fill_function_pointers<<<1, 1>>>(d_pfuncs);
	hipDeviceSynchronize();

	Programs *d_programs = copy_programs_to_gpu(n_programs, programs);

	err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("Error creating programs: %s\n", hipGetErrorString(err));
		return 1;
	}

	std::cout << "Starting kernel" << std::endl;

	int threads = N_THREADS;

	int blocks = std::min((int)(n_programs / threads), N_BLOCKS);

	create_and_run<<<blocks, threads>>>(d_programs, n_programs, problems, d_pfuncs, d_accuracy, blocks, threads);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("Error launching kernel: %s\n", hipGetErrorString(err));
		// Handle the error (e.g., exit the program)
		return 1;
	}
	std::cout << "Kernel finished" << std::endl;

	free_programs_from_gpu(d_programs);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("Error freeing programs: %s\n", hipGetErrorString(err));
		return 1;
	}

	hipMemcpy(accuracy, d_accuracy, n_programs * sizeof(float), hipMemcpyDeviceToHost);

	err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("Error launching kernel: %s\n", hipGetErrorString(err));
		// Handle the error (e.g., exit the program)
		return 1;
	}

	float total = 0.0;

	for (int i = 0; i < n_programs; i++)
	{
		total += accuracy[i];
	}

	std::cout << "Total: " << total << " " << n_programs << std::endl;

	hipFree(d_pfuncs);
	hipFree(d_accuracy);

	return 0;
}

/*
int main()
{
	int n_programs = 30000;
	std::vector<std::string> programs;

	for (int i = 0; i < n_programs; i++)
	{
		programs.push_back("prog2(testing_output_write(get0()),testing_output_move_right())");
		// programs.push_back("get0()");
	}

	float *accuracy;
	accuracy = (float *)malloc(n_programs * sizeof(float));

	execute_and_evaluate(n_programs, programs.data(), accuracy);
}*/