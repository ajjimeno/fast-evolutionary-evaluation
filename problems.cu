
#ifndef PROGRAM_PROBLEMS_C
#define PROGRAM_PROBLEMS_C

Problems *load_problems()
{
    int in[3][3] = {{0, 0, 5},
                    {0, 5, 0},
                    {5, 0, 0}};

    int out[3][3] = {{3, 3, 3},
                     {4, 4, 4},
                     {2, 2, 2}};

    // Example set creation
    Problem problem = {
        (int **)in,  // input
        3,           // input_x
        3,           // input_y
        (int **)out, // out_gt
        (int **)out, // output
        3,           // output_x
        3            // output_y
    };

    int **d_in = (int **)malloc(problem.input_y * sizeof(int *));

    for (int i = 0; i < problem.input_y; i++)
    {
        hipMalloc(&d_in[i], problem.input_x * sizeof(int));
        hipMemcpy(d_in[i], in[i], problem.input_x * sizeof(int), hipMemcpyHostToDevice);
    }

    hipMalloc(&problem.input, problem.input_y * sizeof(int *));
    hipMemcpy(problem.input, d_in, problem.input_y * sizeof(int *), hipMemcpyHostToDevice);

    free(d_in);

    int **d_output = (int **)malloc(problem.output_y * sizeof(int *));

    for (int i = 0; i < problem.output_y; i++)
    {
        hipMalloc(&d_output[i], problem.output_x * sizeof(int));
        hipMemcpy(d_output[i], out[i], problem.output_x * sizeof(int), hipMemcpyHostToDevice);
    }

    hipMalloc(&problem.output, problem.output_y * sizeof(int *));
    hipMemcpy(problem.output, d_in, problem.output_y * sizeof(int *), hipMemcpyHostToDevice);

    free(d_output);

    int **d_output_gt = (int **)malloc(problem.output_y * sizeof(int *));

    for (int i = 0; i < problem.output_y; i++)
    {
        hipMalloc(&d_output_gt[i], problem.output_x * sizeof(int));
        hipMemcpy(d_output_gt[i], out[i], problem.output_x * sizeof(int), hipMemcpyHostToDevice);
    }

    hipMalloc(&problem.out_gt, problem.output_y * sizeof(int *));
    hipMemcpy(problem.out_gt, d_in, problem.output_y * sizeof(int *), hipMemcpyHostToDevice);

    free(d_output_gt);

    Problem *d_problem;
    hipMalloc(&d_problem, sizeof(Problem));
    hipMemcpy(d_problem, &problem, sizeof(Problem), hipMemcpyHostToDevice);

    Problems h_problems;

    h_problems.n_problems = 1;

    hipMalloc(&h_problems.problems, sizeof(Problem));
    hipMemcpy(h_problems.problems, d_problem, sizeof(Problem), hipMemcpyHostToDevice);

    Problems *d_problems;
    hipMalloc(&d_problems, sizeof(Problems));
    hipMemcpy(d_problems, &h_problems, sizeof(Problems), hipMemcpyHostToDevice);

    return d_problems;
}

__device__ float accuracy_calculation(Problem problem, int **output)
{
    float tp = 0.0;

    // Count number of equal entries
    for (int i = 0; i < problem.output_y; i++)
    {
        for (int j = 0; j < problem.output_x; j++)
        {
            if (problem.out_gt[i][j] == output[i][j])
            {
                tp++;
            }
        }
    }

    // Total number of entries
    int total = problem.output_y * problem.output_x;

    return (float)tp / (float)total;
}

#endif